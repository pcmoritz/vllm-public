#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "../dispatch_utils.h"

#include <c10/util/BFloat16.h>
#include <c10/cuda/CUDAStream.h>

#include "cutlass/bfloat16.h"
#include "cutlass/complex.h"
#include "cutlass/gemm/kernel/gemm_grouped.h"
#include "cutlass/gemm/kernel/default_gemm_grouped.h"
#include "cutlass/gemm/device/gemm_grouped.h"

namespace vllm {

#define CUDA_CALL(code)					                    \
  do {                                              \
    hipError_t status = code;                      \
    std::string err = hipGetErrorString(status);   \
    TORCH_CHECK(status == hipSuccess, err);		    \
  } while (0)

using GroupedGemmKernelNN = typename cutlass::gemm::kernel::DefaultGemmGrouped<
  // Non-transposed A operand.
  ::cutlass::bfloat16_t,
  ::cutlass::layout::RowMajor,
  ::cutlass::ComplexTransform::kNone,
  8,
  // Non-transposed B operand.
  ::cutlass::bfloat16_t,
  ::cutlass::layout::RowMajor,
  ::cutlass::ComplexTransform::kNone,
  8,
  // C operand.
  ::cutlass::bfloat16_t,
  ::cutlass::layout::RowMajor,
  float,
  ::cutlass::arch::OpClassTensorOp,
  ::cutlass::arch::Sm80,
  ::cutlass::gemm::GemmShape<128, 128, 32>,
  ::cutlass::gemm::GemmShape<64, 64, 32>,
  ::cutlass::gemm::GemmShape<16, 8, 16>,
  ::cutlass::epilogue::thread::LinearCombination<::cutlass::bfloat16_t, 8, float, float>,
  // NOTE: Threadblock swizzling is currently not supported by CUTLASS's grouped kernels.
  // This parameter is passed in at present to match the APIs of other kernels. The parameter
  // is unused within the kernel.
  ::cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
  4>::GemmKernel;
using GemmGroupedNN = ::cutlass::gemm::device::GemmGrouped<GroupedGemmKernelNN>;


std::vector<cutlass::gemm::GemmCoord> MakeProblemSizes(torch::Tensor b, torch::Tensor cum_num_tokens_per_expert) {
  const size_t num_experts = cum_num_tokens_per_expert.size(0);
  const size_t k = b.size(1), n = b.size(2);
  std::vector<cutlass::gemm::GemmCoord> problem_sizes(num_experts);
  for (int i = 0; i < num_experts; ++i) {
    int64_t batch_size = cum_num_tokens_per_expert.data_ptr<int64_t>()[i] - (i > 0 ? cum_num_tokens_per_expert.data_ptr<int64_t>()[i-1] : 0);
    problem_sizes[i] = cutlass::gemm::GemmCoord(batch_sizes.data_ptr<int64_t>()[i], n, k);
  }
  return problem_sizes;
}

template <typename T>
torch::Tensor CopyToDevice(const std::vector<T> &x, const torch::Device &device) {
  size_t bytes = x.size() * sizeof(T);
  auto options = torch::TensorOptions().dtype(torch::kInt8).device(device);
  torch::Tensor out = torch::empty(bytes, options);

  CUDA_CALL(hipMemcpyAsync(out.data_ptr(),
			    x.data(), bytes,
			    hipMemcpyHostToDevice,
			    c10::cuda::getCurrentCUDAStream()));
  return out;
}

template <typename Gemm>
typename Gemm::Arguments MakeArguments(torch::Tensor a,
				       torch::Tensor b,
				       torch::Tensor c,
				       torch::Tensor batch_sizes) {
  auto problem_sizes_host = MakeProblemSizes(b, batch_sizes);

  // Calculate the number of threadblocks to use and validate the result.
  int64_t num_experts = problem_sizes_host.size();

  // NOTE: This is borrowed from FasterTransformer.
  int threadblock_count = Gemm::sufficient(problem_sizes_host.data(), num_experts);
  if (!threadblock_count) {
    TORCH_CHECK(false, "Grouped GEMM execution not possible with HW");
  }

  // Create the host arrays of leading dimension data and pointer data.
  using LayoutA = typename Gemm::LayoutA;
  using LayoutB = typename Gemm::LayoutB;
  using LayoutC = typename Gemm::LayoutC;

  std::vector<int64_t> lda_host(num_experts), offsets_a(num_experts);
  std::vector<int64_t> ldb_host(num_experts), offsets_b(num_experts);
  std::vector<int64_t> ldc_host(num_experts), offsets_c(num_experts);
  int64_t elements_a = 0, elements_b = 0, elements_c = 0;

  using ElementA = typename Gemm::ElementA;
  using ElementB = typename Gemm::ElementB;
  using ElementC = typename Gemm::ElementC;
  std::vector<ElementA *> ptr_a_host(num_experts);
  std::vector<ElementB *> ptr_b_host(num_experts);
  std::vector<ElementC *> ptr_c_host(num_experts);

  for (int i = 0; i < num_experts; ++i) {
    auto problem = problem_sizes_host[i];
    lda_host[i] = LayoutA::packed({problem.m(), problem.k()}).stride(0);
    ldb_host[i] = LayoutB::packed({problem.k(), problem.n()}).stride(0);
    ldc_host[i] = LayoutC::packed({problem.m(), problem.n()}).stride(0);

    offsets_a[i] = elements_a;
    offsets_b[i] = elements_b;
    offsets_c[i] = elements_c;

    ptr_a_host[i] = (ElementA*)a.data_ptr() + offsets_a[i];
    ptr_b_host[i] = (ElementB*)b.data_ptr() + offsets_b[i];
    ptr_c_host[i] = (ElementC*)c.data_ptr() + offsets_c[i];

    elements_a += problem.m() * problem.k();
    elements_b += problem.k() * problem.n();
    elements_c += problem.m() * problem.n();
  }

  // Copy the problem sizes, pointers and leading dimension data to the device.
  torch::Tensor lda = CopyToDevice(lda_host, a.device());
  torch::Tensor ldb = CopyToDevice(ldb_host, a.device());
  torch::Tensor ldc = CopyToDevice(ldc_host, a.device());
  torch::Tensor ptr_a = CopyToDevice(ptr_a_host, a.device());
  torch::Tensor ptr_b = CopyToDevice(ptr_b_host, a.device());
  torch::Tensor ptr_c = CopyToDevice(ptr_c_host, a.device());
  torch::Tensor problem_sizes = CopyToDevice(problem_sizes_host, a.device());

  typename Gemm::EpilogueOutputOp::Params epilogue_op(/*alpha=*/1.0f, /*beta=*/0.0f);
  typename Gemm::Arguments arguments((cutlass::gemm::GemmCoord*)problem_sizes.data_ptr(),
  				     (int)num_experts,
  				     (int)threadblock_count,
  				     epilogue_op,
  				     (ElementA**)ptr_a.data_ptr(),
  				     (ElementB**)ptr_b.data_ptr(),
  				     (ElementC**)ptr_c.data_ptr(),
  				     (ElementC**)ptr_c.data_ptr(),
  				     /*lda=*/(int64_t*)lda.data_ptr(),
  				     /*ldb=*/(int64_t*)ldb.data_ptr(),
  				     /*ldc=*/(int64_t*)ldc.data_ptr(),
  				     /*ldd=*/(int64_t*)ldc.data_ptr(),
  				     (cutlass::gemm::GemmCoord*)problem_sizes_host.data());
  return arguments;
}

void CutlassGroupedGemm(torch::Tensor a,
				 torch::Tensor b,
				 torch::Tensor c,
				 torch::Tensor cum_num_tokens_per_expert,
         hipStream_t stream) {
  using Gemm = GemmGroupedNN;
  Gemm gemm;

  auto arguments = MakeArguments<Gemm>(a, b, c, cum_num_tokens_per_expert);
  int64_t workspace_size = gemm.get_workspace_size(arguments);
  auto options = torch::TensorOptions().dtype(torch::kInt8).device(a.device());
  torch::Tensor workspace = torch::empty(workspace_size, options);

  // Initialize the kernel.
  if(gemm.initialize(arguments, workspace.data_ptr()) != cutlass::Status::kSuccess) {
    TORCH_CHECK(false, "Failed to initialize CUTLASS Grouped GEMM");
  }

  // Execute the kernel in the current stream.
  if(gemm.run(stream) != cutlass::Status::kSuccess) {
    TORCH_CHECK(false, "Failed to run CUTLASS Grouped GEMM");
  }
}


template <class T, class ActFn>
__global__ void doGatedActivationKernel(
    T* output, const T* gemm_result, const int64_t* num_valid_tokens_ptr, size_t inter_size)
{
    const int tid = threadIdx.x;
    const int token = blockIdx.x;
    if (num_valid_tokens_ptr && token >= *num_valid_tokens_ptr)
    {
        return;
    }

    ActFn fn{};
    output = output + token * inter_size;
    gemm_result = gemm_result + token * inter_size * 2;
    for (int i = tid; i < inter_size; i += blockDim.x)
    {
        T fc1_value = gemm_result[i];
        // BF16 isn't supported, use FP32 for activation function
        float gate_value = gemm_result[i + inter_size];
        T gate_act = fn(gate_value);
        output[i] = fc1_value * gate_act;
    }
}

template <class T>
void doGatedActivation(T* output, const T* gemm_result, const int64_t* num_valid_tokens_ptr, int inter_size,
    int num_tokens, hipStream_t stream)
{
    const int blocks = num_tokens;
    const int threads = std::min(inter_size, 1024);

    // TODO Instead of T use a vectored type if performance would benefit
    // TODO For some reason Volta fails on GELU_taylor here with Warp Illegal Instruction.
    auto* fn = &doGatedActivationKernel<T, cutlass::epilogue::thread::SiLu<float>>;
    fn<<<blocks, threads, 0, stream>>>(output, gemm_result, num_valid_tokens_ptr, inter_size);
}

}  // namespace

void moe_mlp(
    torch::Tensor& moe_output,                              // [num_tokens * topk, hidden_size]
    torch::Tensor& input_tokens,                            // [num_tokens * topk, hidden_size]
    torch::Tensor& cum_num_tokens_per_expert,               // [num_experts]
    torch::Tensor& fc1_expert_weights,                      // [num_experts, 2 * inter_size, hidden_size]
    torch::Tensor& fc2_expert_weights)                      // [num_experts, hidden_size, inter_size]
{
  const int64_t num_expanded_tokens = input_tokens.numel() / input_tokens.size(-1);
  const int num_experts = fc2_expert_weights.size(0);
  const int hidden_size = fc2_expert_weights.size(1);
  const int inter_size = fc2_expert_weights.size(2);

  const at::cuda::OptionalCUDAGuard device_guard(device_of(input_tokens));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  torch::Tensor fc1_output = torch::empty({num_expanded_tokens, inter_size}, input_tokens.options());
  torch::Tensor glu_output = torch::empty({num_expanded_tokens * inter_size * 2}, input_tokens.options());

  vllm::CutlassGroupedGemm(input_tokens, fc1_expert_weights, fc1_output, cum_num_tokens_per_expert, stream);

  vllm::doGatedActivation<__hip_bfloat16>(
    (__hip_bfloat16*) fc1_output.data_ptr<at::BFloat16>(),
    (__hip_bfloat16*) glu_output.data_ptr<at::BFloat16>(),
    nullptr, inter_size, num_expanded_tokens, stream);

  vllm::CutlassGroupedGemm(fc1_output, fc2_expert_weights, moe_output, cum_num_tokens_per_expert, stream);
}
